#define interactionRange2 (1.0) 
#define MaxThreads (256)

#include "activefluid_multi.c"
#include "stdlib.h"
#include <time.h>
#include <random>
#include <fstream>
#include <iostream>
int main(int argc, char *argv[])
{
    if(argc!=10) error_output("command L tmax delt") ;
    time_t start, end;
    const double  rho   = 800.0;
    const int    Lsize  = atoi(argv[1]);
    const int    tmax   = atoi(argv[2]);
    const double dt     = atof(argv[3]);
    const int    record = atof(argv[4]);
    const int    N_body = atoi(argv[5]);
    const double lamb   = atof(argv[6]);
    const double dist   = atof(argv[7]);
    const double mu_R_A = atof(argv[8])/360.0*two_ppi/rho;
    const double mu_R_C = atof(argv[9])/360.0*two_ppi/rho;
    const double mu_active = 1.0;
    const double alpha = 1.0;
    //const int    N_C    = atoi(argv[5]);
    //const int    N_A    = atoi(argv[6]);
    const double  U0  = 1.0;

    // total number of particles
    const  int   N_active = (int)(Lsize*Lsize*rho);
    const  int   N_passive = (int)(Lsize/dist);
    const  int   N_ptcl = N_active+N_passive*N_body*N_passive ;
    const  double dtheta = ppi/N_body;
    // total number of cells
    const  int   cllsNum    = Lsize*Lsize ;

    // grid dimension
    const int nThreads = (MaxThreads<N_ptcl)? MaxThreads : N_ptcl;
    const int nBlocks  = (N_ptcl+nThreads-1)/nThreads;
    std::cout<<N_ptcl<<'\n';
    double *paAngle, *paTorque, *pax, *pay;
    double *AngleHost;
    hipMalloc(&paAngle, sizeof(double)*N_passive*N_passive);
    hipMalloc(&paTorque, sizeof(double)*N_passive*N_passive);
    hipMalloc(&pax, sizeof(double)*N_passive*N_passive);
    hipMalloc(&pay, sizeof(double)*N_passive*N_passive);
    AngleHost = (double *)malloc(sizeof(double)*N_passive*N_passive);
    // VicsekParticle in the device
    struct particle *devPtls;
    hipMalloc(&devPtls, sizeof(struct particle)*N_ptcl) ;
    // auxiliary memory for linked lists
    // linked list is managed with the THRUST library
    // corresponding device memory
    int *devCell, *devHead, *devTail ;
    double *devtorque;
    hipMalloc(&devCell, sizeof(int)*N_ptcl);
    hipMalloc(&devHead, sizeof(int)*cllsNum);
    hipMalloc(&devTail, sizeof(int)*cllsNum);
    hipMalloc(&devtorque, sizeof(double)*(N_passive*N_passive*N_body));
    

/* // temporary angle variable
    float *devAngTmp;
    hipMalloc(&devAngTmp, sizeof(double)*ptlsNum);

    // order parameter measurement
    float *devVx, *devVy;
    hipMalloc(&devVx, sizeof(float)*ptlsNum);
    hipMalloc(&devVy, sizeof(float)*ptlsNum);
*/
    // set the PRNG seed with the device random number
    hiprandState *devStates;
    hipMalloc(&devStates, sizeof(hiprandState)*N_ptcl);
    initrand<<<nBlocks,nThreads>>>(devStates, N_ptcl) ;
        // random initial configuration
        init_object<<<nBlocks,nThreads>>>(devStates,paAngle, pax, pay, N_passive,Lsize,dist);
        init_random_config<<<nBlocks,nThreads>>>(devPtls, devStates, paAngle, pax, pay, Lsize, N_ptcl,N_passive, N_active,N_body,alpha,dtheta) ;
        //init_passive_particle<<<nBlocks,nThreads>>>(devPtls, paAngle, pax, pay, Lsize, N_passive, N_active,N_body,dist) ;
        start = time(NULL);
        std::ofstream out;
        out.open("test.csv");
        for(double t=0.0; t<=tmax; t +=dt) {
            // position and angle update
             // linked list
            linked_list(devPtls, Lsize, N_ptcl,N_active, cllsNum, devCell, devHead, devTail,nBlocks, nThreads);
            force<<<nBlocks,nThreads>>>(devPtls,devHead,devTail,devtorque,pax,pay,Lsize,lamb,N_ptcl,N_passive,N_active,N_body);
            torque_object<<<nBlocks,nThreads>>>(devtorque, paTorque, paAngle,N_passive,N_body,mu_R_A,mu_R_C);
            particles_move<<<nBlocks, nThreads>>>(devPtls,devStates,paAngle,pax,pay,Lsize,U0,dt,alpha,
                N_ptcl,N_passive,N_active,N_body,mu_active,mu_R_A,mu_R_C);
            if((int)t%record==0)
            {
                hipMemcpy(AngleHost,paAngle,sizeof(double)*N_passive*N_passive,hipMemcpyDeviceToHost);
                for(int i = 0; i<(int)N_passive*N_passive/2;i++)
                {
                    out<<AngleHost[i]<<',';
                    out<<AngleHost[i+(int)N_passive*N_passive/2]<<',';
                }
                out<<'\n';
            }
        }
        end= time(NULL);
        std::cout<<(end-start)<<'\n';
    //}

    hipFree(devPtls) ; hipFree(devStates);  
    hipFree(devCell) ; hipFree(devHead) ; hipFree(devTail) ; 
    hipFree(paAngle); hipFree(pax);hipFree(pay);hipFree(paTorque);
    free(AngleHost);
}